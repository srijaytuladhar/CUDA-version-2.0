#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "timer.c"

/*********************************************************
 * 
 * 
 * To Compile:
 * nvcc -o cuda_password_2_initials_4_digits cuda_password_2_initials_4_digits.cu
 * 
 * 
 * To Run:
 * ./cuda_password_2_initials_4_digits
 * 
 * 
 *********************************************************/

/* password_match_checker function */
__device__ int password_match_checker(char *password_trial) {
    
    char assigned_password_1[] = "BV7842";      // 1 of 4 passwords
    char assigned_password_2[] = "ES2107";      // 2 of 4 passwords
    char assigned_password_3[] = "HR2332";      // 3 of 4 passwords
    char assigned_password_4[] = "RB9669";      // 4 of 4 passwords

    char *attempt_1 = password_trial;
    char *attempt_2 = password_trial;
    char *attempt_3 = password_trial;
    char *attempt_4 = password_trial;
    
    char *password_1 = assigned_password_1;
    char *password_2 = assigned_password_2;
    char *password_3 = assigned_password_3;
    char *password_4 = assigned_password_4;

    while(*attempt_1 == *password_1) {
        if(*attempt_1 == '\0')
        {
            printf("%s\n", assigned_password_1);
            break;
        }

        attempt_1++;
        password_1++;
    }
        
    while(*attempt_2 == *password_2) {
        if(*attempt_2 == '\0')
        {
            printf("%s\n", assigned_password_2);
            break;
        }

        attempt_2++;
        password_2++;
    }

    while(*attempt_3 == *password_3) {
        if(*attempt_3 == '\0')
        {
            printf("%s\n", assigned_password_3);
            break;
        }

        attempt_3++;
        password_3++;
    }

    while(*attempt_4 == *password_4) {
        if(*attempt_4 == '\0')
        {
            printf("%s",assigned_password_1);
            return 1;
        }

        attempt_4++;
        password_4++;
    }
    
    return 0;

}

/* kernel_function */
__global__ void  kernel_function() {
    
    char digit_1, digit_2, digit_3, digit_4;
 
    char asg_password[7];         // assigned password with size 7 
    asg_password[6] = '\0';       // for terminating the string
    
    int loop_1 = blockIdx.x+65;
    int loop_2 = threadIdx.x+65;

    char firstValue = loop_1;
    char secondValue = loop_2;
        
    asg_password[0] = firstValue;       // 1st letter of the password
    asg_password[1] = secondValue;      // 2nd letter of the password
    for(digit_1 ='0'; digit_1<='9'; digit_1++){
        for(digit_2 ='0'; digit_2<='9'; digit_2++){
            for(digit_3 ='0'; digit_3<='9'; digit_3++){
                for(digit_4 ='0'; digit_4<='9'; digit_4++){
                    asg_password[2] = digit_1;       // 3rd letter of the password
                    asg_password[3] = digit_2;       // 4th letter of the password
                    asg_password[4] = digit_3;       // 5th letter of the password
                    asg_password[5] = digit_4;       // 6th letter of the password
                    if(password_match_checker(asg_password)) {
                        //printf("\n Password Matched!!");      // declared to print message but is not used
                    }
                }
            } 
        }
    }
}


/* main function */
int main() {

    // for time (start)   
    struct timespec timer_start, timer_stop;   
    long long int time_taken_for_execution;
    
    clock_gettime(CLOCK_MONOTONIC, &timer_start);
    printf("\n===============================================================================\n");
    printf("!! MATCHED PASSWORD !! \n");
    printf("===============================================================================\n\n"); 
    
    kernel_function <<<26,26>>>();
    hipDeviceSynchronize();

    
    // for time (end)
    clock_gettime(CLOCK_MONOTONIC, &timer_stop);
    timer_calc(&timer_start, &timer_stop, &time_taken_for_execution);
    
    // output of time taken for execution  is displayed
    printf("\n\n===============================================================================\n");
    printf("!! TIME TAKEN FOR EXECUTION !! \n");
    printf("===============================================================================\n\n");
    printf("Nanoseconds: %lld\n", time_taken_for_execution); 
    printf("Seconds: %0.9lf\n\n", ((time_taken_for_execution/1.0e9))); 
    //printf("Minutes: %0.4lf\n", ((time_taken_for_execution/1.0e9)/60));
    //printf("Hours: %0.2lf\n\n", ((time_taken_for_execution/1.0e9)/3600)); 
    

    return 0;
}



